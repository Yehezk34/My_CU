#include "hip/hip_runtime.h"
#include "Header_Lib.h"
#include <stdio.h>

#define M 100 // number of times to do the data transfer
#define TPB 32


__device__ float distance(float x1, float x2){
	return sqrt((x2 - x1) *(x2 - x1));
}

__global__ void distanceKernel(float *d_out, float *d_in, float ref){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x = d_in[i];
	d_out[i] = distance(x, ref);
	printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

void distanceArray(float *out, float *in, float ref, int len){
	hipEvent_t startMemcpy, stopMemcpy;
	hipEvent_t startKernel, stopKernel;
	hipEventCreate(&startMemcpy);
	hipEventCreate(&stopMemcpy);
	hipEventCreate(&startKernel);
	hipEventCreate(&stopKernel);

	float *d_in = 0;
	float *d_out = 0;

	hipMalloc(&d_in, len*sizeof(float));
	hipMalloc(&d_out, len*sizeof(float));

	hipEventRecord(startMemcpy);

	for (int i = 0; i < M; ++i){
		hipMemcpy(d_in, in, len*sizeof(float), hipMemcpyHostToDevice);
	}

	hipEventRecord(stopMemcpy);

	hipEventRecord(startKernel);
	distanceKernel<<<len/TPB, TPB>>>(d_out, d_in, ref);
	hipEventRecord(stopKernel);

	hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);
	
	hipEventSynchronize(stopMemcpy);
	hipEventSynchronize(stopKernel);

	float memcpyTimeInMs = 0;
	hipEventElapsedTime(&memcpyTimeInMs, startMemcpy, stopMemcpy);
	float kernelTimeInMs = 0;
	hipEventElapsedTime(&kernelTimeInMs, startKernel, stopKernel);	

	printf("kernel time (ms): %f\n", kernelTimeInMs);
	printf("data transfer time (ms): %f\n", memcpyTimeInMs);	

	hipFree(d_in);
	hipFree(d_out);
}